
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
// function to add the elements of two array
void add(int n, float *x, float *y)
{
  // get the index of the thread inside the block
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// as well as the number of threads per block
	int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements
  std::cout << "Num elements: " << N << std::endl;

  // allocate unified memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  std::cout << " num blocks: " << numBlocks << std::endl;
  // Run kernel on 1M elements on the CPU
  add<<<numBlocks, blockSize>>>(N, x, y);

  // wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}